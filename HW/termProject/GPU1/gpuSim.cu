
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
//#include "mass.cu"

unsigned int N;
unsigned int MASSES_PER_CORE;

double scientificNotation(double num,  int exp)
{ 
	if(exp > 0)
	{
		for(int i=0; i<exp; i++){
			num *= 10; 
		}
	}else{
		for(int i=0; i>exp; i--){
			num /= 10; 
		}
	}
	return num; 
}

double G = 0;

void initG(){
	G = scientificNotation(6.67384, -11);
}

typedef struct Mass
{
	double objectMass;
	double positionX;
	double positionY;
	double positionZ;
	double velocityX;
	double velocityY;
	double velocityZ;
	double cumalForcesX;
	double cumalForcesY;
	double cumalForcesZ;
}Mass;

__device__
void resetForces(Mass *m){
	m->cumalForcesX = 0;
	m->cumalForcesY = 0;
	m->cumalForcesZ = 0;
}

__device__
double newtonGrav(double m1Mass, double m2Mass, double distance, double localG){
	return localG*(m1Mass*m2Mass)/(distance*distance);
}

__device__
void influence(Mass *m1, Mass *m2, double localG){
	double diffPosX = m1->positionX - m2->positionX;
	double diffPosY = m1->positionY - m2->positionY;
	double diffPosZ = m1->positionZ - m2->positionZ;
	double distance = sqrt(diffPosX*diffPosX + diffPosY*diffPosY + diffPosZ*diffPosZ);
	//*dist = distance; 
	//*dist = 15; 
	double netForce = newtonGrav(m1->objectMass, m2->objectMass, distance, localG);
	//*dist = netForce;
	m1->cumalForcesX += netForce * diffPosX/distance; 
	m1->cumalForcesY += netForce * diffPosY/distance; 
	m1->cumalForcesZ += netForce * diffPosZ/distance;
}

__device__
void updateVelAndPos(Mass *m, double timeStep){
	double accelerationX = m->cumalForcesX/m->objectMass;
	double accelerationY = m->cumalForcesY/m->objectMass;
	double accelerationZ = m->cumalForcesZ/m->objectMass;
	double timeStepSquared = timeStep*timeStep;

	m->positionX += m->velocityX*timeStep + 0.5*(accelerationX)*(timeStepSquared);
	//*dist = m->velocityX*timeStep + 0.5*(accelerationX)*(timeStepSquared);
	m->positionY += m->velocityY*timeStep + 0.5*(accelerationY)*(timeStepSquared);
	m->positionZ += m->velocityZ*timeStep + 0.5*(accelerationZ)*(timeStepSquared);

	m->velocityX += (accelerationX)*(timeStep);
	m->velocityY += (accelerationY)*(timeStep);
	m->velocityZ += (accelerationZ)*(timeStep);
}

__global__
void simulate(Mass * masses, unsigned long numMasses, unsigned int massesPerBlock, double deltaT, unsigned long totalTimeSteps, double localG)
{
	unsigned int myId = blockIdx.x*massesPerBlock + threadIdx.x; 
	for(unsigned int i=0; i<totalTimeSteps; i++)
	{
		// Sync threads so positions are not updated before all other 
		__syncthreads(); 

		// Calc forces on all masses
		for(unsigned long i=0; i<numMasses; i++){
			if(i != myId)
				influence(&masses[myId], &masses[i], localG); 
		}
		

		// Sync threads so positions are not updated before all other 
		__syncthreads(); 

		// Update position of all masses
		updateVelAndPos(&masses[myId], deltaT); 

		// Reset forces
		resetForces(&masses[myId]);
	} 
}

__global__
void testEff(Mass * masses)//, unsigned long numMasses, double deltaT, unsigned long totalTimeSteps, double localG)
{
	masses[threadIdx.x].positionX = (double)threadIdx.x; 
}

__global__
void testInfluence(Mass * masses, unsigned int numMasses, double localG){
	for(unsigned long i=0; i<numMasses; i++)
	{
		if(i != threadIdx.x)
		{
			influence(&masses[threadIdx.x], &masses[i], localG); 
		}
	}
}


int main(int argc, char ** argv)
{
	initG(); 

	// Simulation parameter variables
	double TIME_STEP_SIZE = 1;
	unsigned long TOTAL_SIM_STEPS = 1000;

	// Custom simulation parameters 
	if(argc > 1)
	{
		if(argc != 5){
			std::cout << std::endl << "ERROR, incorrect number of arguments" << std::endl; 
			return -1; 
		}else{
			TOTAL_SIM_STEPS = atoi(argv[1]); 
			TIME_STEP_SIZE = atoi(argv[2]); 
			N = atoi(argv[3]); 
			MASSES_PER_CORE = atoi(argv[4]); 
		}
	}
	std::cout << "Simulation: " << std::endl << "Number of steps = " << TOTAL_SIM_STEPS 
		<< std::endl << "Size of time step (seconds) = " << TIME_STEP_SIZE
		<< std::endl << "N (number of masses) = " << N
		<< std::endl << "Masses per core = " << MASSES_PER_CORE 
		<< std::endl << "Value of G = " << G << std::endl;
 

	// Make masses	
	Mass * h_massArray =(Mass*) malloc(N*sizeof(Mass)); 
	
	// Populate array of masses
	for(int i=0; i<N; i++){
		h_massArray[i].objectMass = scientificNotation(6+i, 23); 
		h_massArray[i].positionX = scientificNotation(6+i, 10); 	
		h_massArray[i].positionY = scientificNotation(6+i, 10); 	
		h_massArray[i].positionZ = scientificNotation(6+i, 10); 	
		h_massArray[i].velocityX = scientificNotation(6+i, 1); 	
		h_massArray[i].velocityY = scientificNotation(6+i, 2); 	
		h_massArray[i].velocityZ = scientificNotation(6+i, 1); 	
		h_massArray[i].cumalForcesX = 0; 	
		h_massArray[i].cumalForcesY = 0; 	
		h_massArray[i].cumalForcesZ = 0; 	
	}

	// Start output
	std::cout << "Start Posisions (X): " << std::endl;
	//for(int i=0; i<N; i++){
	//	std::cout << h_massArray[i].positionX << std::endl; 
	//}
	std::cout << h_massArray[0].positionX << std::endl; 


	// Allocate memory on device for masses
	Mass * d_massArray;
	hipMalloc( (void**)&d_massArray, N*sizeof(Mass));


	// Copy masses onto device
	hipMemcpy( d_massArray, h_massArray, (N*sizeof(Mass)), hipMemcpyHostToDevice );

	// Dimensions for cuda function call 
	dim3 blockDimensions( 32, 1 );
	dim3 gridDimensions( 8, 1 );

	// Do simi
	unsigned int massesPerBlock = N/blockDimensions.x; 
	//unsigned int massesPerThread = massesPerBlock/gridDimensions.x;
 
	simulate<<< gridDimensions, blockDimensions >>>(d_massArray, N, massesPerBlock, TIME_STEP_SIZE, TOTAL_SIM_STEPS, G);
	//simulate<<< gridDimensions, blockDimensions >>>(d_massArray, N, MASSES_PER_CORE, TIME_STEP_SIZE, TOTAL_SIM_STEPS, G);
	//testEff<<< gridDimensions, blockDimensions >>>(d_massArray);
	//testInfluence<<< gridDimensions, blockDimensions >>>(d_massArray, N, G, d_dist); 

	// Get data back
	hipMemcpy( h_massArray, d_massArray, (N*sizeof(Mass)), hipMemcpyDeviceToHost );

	// Free device mem 
	hipFree( d_massArray );

	// Output
	std::cout << std::endl << "End Posisions (X): " << std::endl;
	//std::cout << "Distance: " << h_dist << std::endl;
	//for(int i=0; i<N; i++){
	//	std::cout << h_massArray2[i].positionX << std::endl; 
	//	std::cout << h_massArray2[i].cumalForcesX << std::endl; 	
	//}
	std::cout << h_massArray[0].positionX << std::endl; 

	return EXIT_SUCCESS;
}
