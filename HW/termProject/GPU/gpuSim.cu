
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
//#include "mass.cu"

const int N = 7;

double scientificNotation(double num,  int exp)
{ 
	if(exp > 0)
	{
		for(int i=0; i<exp; i++){
			num *= 10; 
		}
	}else{
		for(int i=0; i>exp; i--){
			num /= 10; 
		}
	}
	return num; 
}

double G = 0;

void initG(){
	G = scientificNotation(6.67384, -11);
}

typedef struct Mass
{
	double objectMass;
	double positionX;
	double positionY;
	double positionZ;
	double velocityX;
	double velocityY;
	double velocityZ;
	double cumalForcesX;
	double cumalForcesY;
	double cumalForcesZ;
}Mass;

__device__
void resetForces(Mass *m){
	m->cumalForcesX = 0;
	m->cumalForcesY = 0;
	m->cumalForcesZ = 0;
}

__device__
double newtonGrav(double m1Mass, double m2Mass, double distance, double localG){
	return localG*(m1Mass*m2Mass)/(distance*distance);
}

__device__
void influence(Mass *m1, Mass *m2, double localG){
	double diffPosX = m1->positionX - m2->positionX;
	double diffPosY = m1->positionY - m2->positionY;
	double diffPosZ = m1->positionZ - m2->positionZ;
	double distance = sqrt(diffPosX*diffPosX + diffPosY*diffPosY + diffPosZ*diffPosZ);
	double netForce = newtonGrav(m1->objectMass, m2->objectMass, distance, localG);
	m1->cumalForcesX += netForce * diffPosX/distance; 
	m1->cumalForcesY += netForce * diffPosY/distance; 
	m1->cumalForcesZ += netForce * diffPosZ/distance;
}

__device__
void updateVelAndPos(Mass *m, double timeStep){
	double accelerationX = m->cumalForcesX/m->objectMass;
	double accelerationY = m->cumalForcesY/m->objectMass;
	double accelerationZ = m->cumalForcesZ/m->objectMass;
	double timeStepSquared = timeStep*timeStep;

	m->positionX += m->velocityX*timeStep + 0.5*(accelerationX)*(timeStepSquared);
	m->positionY += m->velocityY*timeStep + 0.5*(accelerationY)*(timeStepSquared);
	m->positionZ += m->velocityZ*timeStep + 0.5*(accelerationZ)*(timeStepSquared);

	m->velocityX += (accelerationX)*(timeStep);
	m->velocityY += (accelerationY)*(timeStep);
	m->velocityZ += (accelerationZ)*(timeStep);
}

__global__
void simulate(Mass * masses, unsigned long numMasses, double deltaT, unsigned long totalTimeSteps, double localG)
{
	for(unsigned long i=0; i<totalTimeSteps; i++)
	{
		// Sync threads so positions are not updated before all other 
		__syncthreads(); 

		// Calc forces on all masses
		for(unsigned long i=0; i<numMasses; i++)
		{
			if(i != threadIdx.x)
			{
				influence(&masses[threadIdx.x], &masses[i], localG); 
			}
		}

		// Sync threads so positions are not updated before all other 
		__syncthreads(); 

		// Update position of all masses
		updateVelAndPos(&masses[threadIdx.x], deltaT); 

		// Reset forces
		resetForces(&masses[threadIdx.x]);
	} 
}

__global__
void testEff(Mass * masses, unsigned long numMasses, double deltaT, unsigned long totalTimeSteps, double localG)
{
	masses[threadIdx.x].positionX = threadIdx.x; 
}


int main(int argc, char ** argv)
{
	initG(); 

	// Simulation parameter variables
	double TIME_STEP_SIZE = 1;
	unsigned long TOTAL_SIM_STEPS = 1000;

	// Custom simulation parameters 
	if(argc > 1)
	{
		if(argc != 3){
			std::cout << std::endl << "ERROR, incorrect number of arguments" << std::endl; 
			return -1; 
		}else{
			TOTAL_SIM_STEPS = atoi(argv[1]); 
			TIME_STEP_SIZE = atoi(argv[2]); 
		}
	}
	std::cout << "Simulation: " << std::endl << "Number of steps = " << TOTAL_SIM_STEPS 
		<< std::endl << "Size of time step (seconds) = " << TIME_STEP_SIZE << std::endl;
 

	// Make masses	
	Mass * h_massArray =(Mass*) malloc(N*sizeof(Mass)); 
	
	// Populate array of masses
	for(int i=0; i<N; i++){
		h_massArray[i].objectMass = scientificNotation(6+i, 23); 
		h_massArray[i].positionX = scientificNotation(6+i, 10); 	
		h_massArray[i].positionY = scientificNotation(6+i, 10); 	
		h_massArray[i].positionZ = scientificNotation(6+i, 10); 	
		h_massArray[i].velocityX = scientificNotation(6+i, 1); 	
		h_massArray[i].velocityY = scientificNotation(6+i, 2); 	
		h_massArray[i].velocityZ = scientificNotation(6+i, 1); 	
		h_massArray[i].cumalForcesX = 0; 	
		h_massArray[i].cumalForcesY = 0; 	
		h_massArray[i].cumalForcesZ = 0; 	
	}

	// Start output
	for(int i=0; i<N; i++){
		std::cout << h_massArray[i].positionX << std::endl; 
	}


	// Allocate memory on device for masses
	Mass * d_massArray;
	hipMalloc( (void**)&d_massArray, N*sizeof(Mass));

	// Copy masses onto device
	hipMemcpy( d_massArray, h_massArray, (N*sizeof(Mass)), hipMemcpyHostToDevice );

	// Dimensions for cuda function call 
	dim3 blockDimensions( N, 1 );
	dim3 gridDimensions( 1, 1 );

	// Do sim
	//simulate<<< gridDimensions, blockDimensions >>>(d_massArray, N, TIME_STEP_SIZE, TOTAL_SIM_STEPS, G);
	testEff<<< gridDimensions, blockDimensions >>>(d_massArray, N, TIME_STEP_SIZE, TOTAL_SIM_STEPS, G);

	// Get data back
	hipMemcpy( h_massArray, d_massArray, (N*sizeof(Mass)), hipMemcpyDeviceToHost );

	// Free device mem 
	hipFree( d_massArray );

	// Output
	for(int i=0; i<N; i++){
		std::cout << h_massArray[i].positionX << std::endl; 
	}

	return EXIT_SUCCESS;
}
